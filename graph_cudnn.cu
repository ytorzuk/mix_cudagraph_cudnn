#include "hip/hip_runtime.h"
#include "graph_cudnn.h"
#include "cudnn_activation.h"
#include "errors.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>

#define VERBOSE true

__global__ void addKernel(float* c, const float* a, const float* b, unsigned int size)
{
    int i = (1 + blockIdx.x) * threadIdx.x;
    c[i] = a[i] + b[i];
}


hipError_t buildAndRunCudaGraph(float* output, const float* input, const float* bias, unsigned int size)
{
    hipStream_t streamForGraph;
    hipdnnHandle_t cudnn;
    checkCudaErrors(hipStreamCreateWithFlags(&streamForGraph, hipStreamNonBlocking));
    checkCUDNN(hipdnnCreate(&cudnn));
    checkCUDNN(hipdnnSetStream(cudnn, streamForGraph));

    // Original
    float* dev_input = nullptr;
    float* dev_activation = nullptr;
    float* dev_bias = nullptr;
    float* dev_output = nullptr;
    hipError_t cudaStatus;
    int threads = std::min(256u, size);
    int blocks = (size + threads - 1) / threads;
    // For Graph
    hipGraph_t graph;
    std::vector<hipGraphNode_t> nodeDependencies;
    hipGraphNode_t memcpyInputNode, memcpyBiasNode, memcpyOutputNode, kernelNode, cudnnNode;
    hipKernelNodeParams kernelNodeParams = { 0 };
    hipHostNodeParams cudnnNodeParams = { 0 };
    hipMemcpy3DParms memcpyParams = { 0 };
    // Choose which GPU to run on, change this on a multi-GPU system. Then allocate GPU memory.
    {
        cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) {
            std::cerr << "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n";
        }
        cudaStatus = hipMalloc((void**)&dev_output, size * sizeof(float));
        if (cudaStatus != hipSuccess) {
            std::cerr << "hipMalloc failed!\n";
        }
        cudaStatus = hipMalloc((void**)&dev_input, size * sizeof(float));
        if (cudaStatus != hipSuccess) {
            std::cerr << "hipMalloc failed!\n";
        }
        cudaStatus = hipMalloc((void**)&dev_bias, size * sizeof(float));
        if (cudaStatus != hipSuccess) {
            std::cerr << "hipMalloc failed!\n";
        }
        cudaStatus = hipMalloc((void**)&dev_activation, size * sizeof(float));
        if (cudaStatus != hipSuccess) {
            std::cerr << "hipMalloc failed!\n";
        }
    }
    // Start of Graph Creation
    checkCudaErrors(hipGraphCreate(&graph, 0));
    // Add memcpy nodes for copying input vectors from host memory to GPU buffers
    memset(&memcpyParams, 0, sizeof(memcpyParams));
    memcpyParams.srcArray = NULL;
    memcpyParams.srcPos = make_hipPos(0, 0, 0);
    memcpyParams.srcPtr = make_hipPitchedPtr((void*)input, size * sizeof(float), size, 1);
    memcpyParams.dstArray = NULL;
    memcpyParams.dstPos = make_hipPos(0, 0, 0);
    memcpyParams.dstPtr = make_hipPitchedPtr(dev_input, size * sizeof(float), size, 1);
    memcpyParams.extent = make_hipExtent(size * sizeof(float), 1, 1);
    memcpyParams.kind = hipMemcpyHostToDevice;
    checkCudaErrors(hipGraphAddMemcpyNode(&memcpyInputNode, graph, NULL, 0, &memcpyParams));
//    hipMemcpy(dev_input, input, size * sizeof(float), hipMemcpyHostToDevice);

    memset(&memcpyParams, 0, sizeof(memcpyParams));
    memcpyParams.srcArray = NULL;
    memcpyParams.srcPos = make_hipPos(0, 0, 0);
    memcpyParams.srcPtr = make_hipPitchedPtr((void*)bias, size * sizeof(float), size, 1);
    memcpyParams.dstArray = NULL;
    memcpyParams.dstPos = make_hipPos(0, 0, 0);
    memcpyParams.dstPtr = make_hipPitchedPtr(dev_bias, size * sizeof(float), size, 1);
    memcpyParams.extent = make_hipExtent(size * sizeof(float), 1, 1);
    memcpyParams.kind = hipMemcpyHostToDevice;
        
    checkCudaErrors(hipGraphAddMemcpyNode(&memcpyBiasNode, graph, NULL, 0, &memcpyParams));
    nodeDependencies.push_back(memcpyBiasNode);

    // Add a cudnn node for launching a kernel on the GPU
    ActivationParams act_params { dev_input, dev_activation, size, cudnn };
//    activation(&act_params);
    memset(&cudnnNodeParams, 0, sizeof(cudnnNodeParams));
    cudnnNodeParams.fn = reinterpret_cast<hipHostFn_t>(activation);
    cudnnNodeParams.userData = &act_params;
    checkCudaErrors(hipGraphAddHostNode(&cudnnNode, graph, &memcpyInputNode, 1, &cudnnNodeParams));
    nodeDependencies.push_back(cudnnNode);

    // Add a kernel node for launching a kernel on the GPU
    memset(&kernelNodeParams, 0, sizeof(kernelNodeParams));
    kernelNodeParams.func = (void*)addKernel;
    kernelNodeParams.gridDim = dim3(blocks, 1, 1);
    kernelNodeParams.blockDim = dim3(threads, 1, 1);
    kernelNodeParams.sharedMemBytes = 0;
    void* kernelArgs[4] = { (void*)&dev_output, (void*)&dev_activation, (void*)&dev_bias, &size };
    kernelNodeParams.kernelParams = kernelArgs;
    kernelNodeParams.extra = NULL;
    checkCudaErrors(hipGraphAddKernelNode(&kernelNode, graph, nodeDependencies.data(), nodeDependencies.size(), &kernelNodeParams));
    nodeDependencies.clear();
    nodeDependencies.push_back(kernelNode);

    // Add memcpy node for copying output vector from GPU buffers to host memory
    memset(&memcpyParams, 0, sizeof(memcpyParams));
    memcpyParams.srcArray = NULL;
    memcpyParams.srcPos = make_hipPos(0, 0, 0);
    memcpyParams.srcPtr = make_hipPitchedPtr(dev_output, size * sizeof(float), size, 1);
    memcpyParams.dstArray = NULL;
    memcpyParams.dstPos = make_hipPos(0, 0, 0);
    memcpyParams.dstPtr = make_hipPitchedPtr(output, size * sizeof(float), size, 1);
    memcpyParams.extent = make_hipExtent(size * sizeof(float), 1, 1);
    memcpyParams.kind = hipMemcpyDeviceToHost;
    checkCudaErrors(hipGraphAddMemcpyNode(&memcpyOutputNode, graph, nodeDependencies.data(), nodeDependencies.size(), &memcpyParams));
    if (VERBOSE) {
        hipGraphNode_t* nodes = NULL;
        size_t numNodes = 0;
        checkCudaErrors(hipGraphGetNodes(graph, nodes, &numNodes));
        std::cout << "Num of nodes in the graph created manually " << numNodes << '\n';
    }
    // Create an executable graph from a graph
    hipGraphExec_t graphExec;
    checkCudaErrors(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0));
    // Run the graph
    checkCudaErrors(hipGraphLaunch(graphExec, streamForGraph));
    checkCudaErrors(hipStreamSynchronize(streamForGraph));
    // Clean up
    checkCudaErrors(hipGraphExecDestroy(graphExec));
    checkCudaErrors(hipGraphDestroy(graph));
    checkCudaErrors(hipStreamDestroy(streamForGraph));
    hipdnnDestroy(cudnn);
    hipFree(dev_output);
    hipFree(dev_input);
    hipFree(dev_bias);
    hipFree(dev_activation);
    return cudaStatus;
}
